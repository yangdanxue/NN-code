#include "hip/hip_runtime.h"
/*
*【当前版本进度】202208051529更新：更新代码。
*/

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
//#include <stdbool.h> 
#include <unistd.h>	//sleep();的库函数
#include "book.cuh"

#define N 						784
#define M						10
#define TestNumber				60000		//全部的图片数

#define CheckNumber				60000		//验证的图片数
#define GPUcalculateNumber			1000		//调用一次核函数计算的图片数
#define pixels					28		//图片像素是28*28

#define OldWeightPercentage			0.9		//旧权重占比
#define NewWeightPercentage			0.1		//新权重占比
#define LearningRate				0.005		//学习速率

int ReadTrainPicData(unsigned char*, unsigned char*);
int ReadTest10kPicData(unsigned char*, unsigned char*); 	

void RandomInitWeight(float*, float*);		

void WriteWeightData(float*, float*);
void ReadWeightInputToHiddenData(float*);			
void ReadWeightHiddenToOutputData(float*);		


void TypeCasting();
void AllocatePointer();
void RunCUDA(int);

void RecognizeResult();
void JudgeRightOrWrong(int, int);

unsigned char *pPicture;								//指向读取的60000张图片数组的一维指针，的主机副本，与，设备副本
float *pNetInput60000,			*d_pNetInput60000;			//指向CPU上开的一个float型的60000张图片大小的数组空间的一维指针（开空间用，指针始终指向空间首地址）
float *pWeightInputToHidden,		*d_pWeightInputToHidden;		
float *pNetInput,				*d_pNetInput;				//指向每次只计算1张图片数组至GPU的指针，的主机副本，与，设备副本（因为是用于定位当前计算的图片的指针，无需开空间+释放空间）
float *pNetHidden,				*d_pNetHidden;			
float *pSigmoidHidden,			*d_pSigmoidHidden;			
float *pWeightHiddenToOutput,		*d_pWeightHiddenToOutput;			
float *pNetOutput,				*d_pNetOutput;			
float *pSigmoidOutput60000,			*d_pSigmoidOutput60000;		//需回传SigmoidOutput到CPU，判断图片是否识别正确。因此，60000张图片的SigmoidOutput数据需全部保留						
float *pSigmoidOutput,			*d_pSigmoidOutput;			
float *pTarget,				*d_pTarget;				 
float *pDelta_NetO,				*d_pDelta_NetO;			
float *pTempPD_NetOToNetH,			*d_pTempPD_NetOToNetH;		
float *pDelta_NetH,				*d_pDelta_NetH;			
unsigned char *pLabel60000,			*d_pLabel60000;			//指向读取的60000张Label数组的一维指针（开空间用，指针始终指向空间首地址），的主机副本，与，设备副本
unsigned char *pLabel,			*d_pLabel;				//指向每次只计算1张图片对应的标签进行计算的GPU的指针，的主机副本，与，设备副本（因为是用于定位当前计算的图片的指针，无需开空间+释放空间）

int iPicNum;		//图片序号
int Recognize;	//图片被识别为数字几
int correct = 0;
int MaxCorrect = 0;
float CorrectRate = 0;




__device__ int d_iNum; //调用第iNum次核函数，每次调用计算1000张图

__device__ __forceinline__ float sigmoid (float a)
{
    return 1.0 / (1.0 + exp (-a));
}

__global__ void ForwardAndBackPropagation(float *pNetInput60000, float *pNetInput, float *pWeightInputToHidden, float *pNetHidden, float *pSigmoidHidden, float *pWeightHiddenToOutput, float *pNetOutput, float *pSigmoidOutput60000, float *pSigmoidOutput, float *pTarget, float *pDelta_NetO, float *pTempPD_NetOToNetH, float *pDelta_NetH, unsigned char *pLabel60000, unsigned char *pLabel)
{
	/******************** GPU上开的整块内存空间中的指针定位分配 ********************/	
	pWeightInputToHidden = 	pNetInput60000 + N*TestNumber;
	pWeightHiddenToOutput = 	pWeightInputToHidden + N*N;
	pNetHidden = 		pWeightHiddenToOutput + N*M;
	pSigmoidHidden = 		pNetHidden + N;					
	pNetOutput = 		pSigmoidHidden + N;					
	pSigmoidOutput60000 = 	pNetOutput + M;					
	pTarget = 			pSigmoidOutput60000 + M*TestNumber; 		   		
	pDelta_NetO = 		pTarget + M; 		
	pTempPD_NetOToNetH = 	pDelta_NetO + M;					
	pDelta_NetH = 		pTempPD_NetOToNetH + N*M;					
	pLabel60000 =		(unsigned char*)(pDelta_NetH + N);

	int X = threadIdx.x;
	
	for(int j = 0; j < GPUcalculateNumber; j++)
	{
		pNetInput  = 	pNetInput60000 +		N*(d_iNum + j);
		pSigmoidOutput = 	pSigmoidOutput60000 +	M*(d_iNum + j);
		pLabel = 		pLabel60000 +		(d_iNum + j);

		float tmpSum = 0;

	  	if (X < N) 
		{
	     		for (int i = 0; i < N; i++) 
			{
		   		tmpSum += pWeightInputToHidden[X * N + i] * pNetInput[i];
			}
	     		pNetHidden[X] = tmpSum;
		    	pSigmoidHidden[X] = sigmoid (pNetHidden[X]);
	     	}
	   	__syncthreads();

		tmpSum = 0;

		if (X < M) 
		{
			for (int i = 0; i < N; i++)
			{
		   		tmpSum += pWeightHiddenToOutput[X * N + i] * pSigmoidHidden[i];
			}
	    		pNetOutput[X] = tmpSum;
 		    	pSigmoidOutput[X] = sigmoid (pNetOutput[X]);
	    	}
		__syncthreads();
	
		if (X < M) 
		{
			for(int i = 0; i < M; i++)
			{
				*(pTarget + i) = 0;	//目标值数组清0
			}
			*(pTarget + *pLabel) = 1;	//标签的值设为1
			pDelta_NetO[X] = -(pTarget[X] - pSigmoidOutput[X]) * pSigmoidOutput[X] * (1 - pSigmoidOutput[X]);
		}
		__syncthreads();

		if (X < N) 
		{
	 	     	for (int i = 0; i < M; i++) 
			{
				pWeightHiddenToOutput[i * N + X] = (pWeightHiddenToOutput[i * N + X])*OldWeightPercentage + 
									    (pWeightHiddenToOutput[i * N + X] - LearningRate * pDelta_NetO[i] * pSigmoidHidden[X])*NewWeightPercentage;
			}
		}
		__syncthreads();

		tmpSum = 0;

		if (X < N) 
		{
	 	     	for (int i = 0; i < M; i++) 
			{
				pTempPD_NetOToNetH[i * N + X] = pWeightHiddenToOutput[i * N + X] * pSigmoidHidden[X] * (1 - pSigmoidHidden[X]);

		   		tmpSum += pTempPD_NetOToNetH[i * N + X] * pDelta_NetO[i];
			}
	    	    	pDelta_NetH[X] = tmpSum;
		}
		__syncthreads();

	    	if (X < N) 
		{
		     	for (int i = 0; i < N; i++) 
			{
				pWeightInputToHidden[X * N + i] = OldWeightPercentage * pWeightInputToHidden[X * N + i] +
									   NewWeightPercentage * (pWeightInputToHidden[X * N + i] - LearningRate * pNetInput[i] * pDelta_NetH[X]);
			}
		}
		__syncthreads();
	}
}
//*****************************************	以上为GPU上运行的代码	*****************************************//






int main(void) 
{
	/******************** 为设备副本分配空间 ********************/
	HANDLE_ERROR( hipMalloc((void **)&d_pNetInput60000,	sizeof(float)*48287718) );

	/******************** 为主机副本分配空间 ********************/
	pPicture		= (unsigned char*)malloc(N*TestNumber);		//从硬盘读到内存上的60000张图片
	pNetInput60000	= (float*)malloc(sizeof(float)*48287718);

	/******************** CPU上开的整块内存中的分配用于定位的指针 ********************/	
	AllocatePointer();

	/******************** 读图片和标签 ********************/
	ReadTrainPicData(pPicture, pLabel60000);
//	ReadTest10kPicData(pPicture, pLabel60000);

	/******************** 读权重or随机初始化权重 ********************/
//	ReadWeightInputToHiddenData(pWeightInputToHidden);
//	ReadWeightHiddenToOutputData(pWeightHiddenToOutput);
	RandomInitWeight(pWeightInputToHidden, pWeightHiddenToOutput);

	/******************** 强制类型转换 ********************/
	TypeCasting();

	/******************** 将输入复制到设备 ********************/
	HANDLE_ERROR( hipMemcpy(d_pNetInput60000, pNetInput60000, sizeof(float)*48287718, hipMemcpyHostToDevice) );

	/******************** 进入网络训练 ********************/
	MaxCorrect = 0;
	for (int traintimes = 0; traintimes < 10; traintimes++)
	{
	 	correct = 0;
		CorrectRate = 0;
		
		printf("第%d次训练：\n", traintimes+1);
		RunCUDA(CheckNumber);

		//******************** (1)判断图片被识别为数字几并判断识别正误(2)分类 ********************
		for (iPicNum = 0; iPicNum < CheckNumber; iPicNum++)
		{
			pSigmoidOutput = 	pSigmoidOutput60000 + 	M*iPicNum;
			pLabel = 		pLabel60000 + 		iPicNum;

			RecognizeResult();
			JudgeRightOrWrong(iPicNum, Recognize);
		}
		CorrectRate = ((float)correct) / CheckNumber;
		printf("正确数：%d 正确率：%.4f%%\n", correct, CorrectRate * 100);
		if(correct > MaxCorrect)
		{
			MaxCorrect = correct;
//			WriteWeightData(pWeightInputToHidden, pWeightHiddenToOutput);
		}
	}

	/******************** 释放主机与设备的内存空间 ********************/
	free(pPicture);
	free(pNetInput60000);
	HANDLE_ERROR( hipFree(d_pNetInput60000) ); 
}


//*****************************************	以下为待调用的函数	*****************************************//
/******************** CPU上开的整块内存中的指针定位分配 ********************/
void AllocatePointer()
{
	pWeightInputToHidden = 	pNetInput60000 + N*TestNumber;
	pWeightHiddenToOutput = 	pWeightInputToHidden + N*N;
	pNetHidden = 		pWeightHiddenToOutput + N*M;
	pSigmoidHidden = 		pNetHidden + N;					
	pNetOutput = 		pSigmoidHidden + N;					
	pSigmoidOutput60000 = 	pNetOutput + M;					
	pTarget = 			pSigmoidOutput60000 + M*TestNumber; 		   		
	pDelta_NetO = 		pTarget + M; 		
	pTempPD_NetOToNetH = 	pDelta_NetO + M;					
	pDelta_NetH = 		pTempPD_NetOToNetH + N*M;					
	pLabel60000 = 		(unsigned char*)(pDelta_NetH + N);	//从float型的指针pDelta_NetH，后移N个float型内存空间（4*N个字节）后的地址，与unsigned char型的指针pLabel60000指向的是同一地址
}

/******************** 判断图片是否被正确识别为对应的label ********************/
void RecognizeResult()
{
	for(int i = 0; i < 10; i++)
	{
		//通过比较SigmoidOutput的大小，得到的最大值即为认定图片被网络识别成几，再与Label比较，看是否识别正确		
		if(*(pSigmoidOutput + i) > *(pSigmoidOutput + Recognize))
		{
			Recognize = i;
		}
	}
}

/******************** 判断图片识别正确或错误 ********************/
void JudgeRightOrWrong(int iNum, int OCR)
{
	if(OCR == *pLabel)
	{
		correct++;
		printf("第%d张图片被正确识别为%d\n", iNum, *pLabel);
	}
	else
	{
		printf("第%d张图片应为%d被错误识别为%d\n", iNum, *pLabel, OCR);
	}
}


/******************** 启用CUDA ********************/
void RunCUDA(int iCheck)
{
	//防止全局指针pNetInput的值一直在累加，跑出数据的有效范围
	pNetInput = 0;
	pSigmoidOutput = 0;
	pLabel = 0;

	for(int iNum = 0; iNum < iCheck; iNum += GPUcalculateNumber)
	{
		HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(d_iNum), &iNum, sizeof(int)) );
		/******************** 在带有N个线程的GPU上启动ForwardAndBackPropagation（）内核 ********************/
		ForwardAndBackPropagation<<<1, N>>>(d_pNetInput60000, d_pNetInput, d_pWeightInputToHidden, d_pNetHidden, d_pSigmoidHidden, d_pWeightHiddenToOutput, d_pNetOutput, d_pSigmoidOutput60000, d_pSigmoidOutput, d_pTarget, d_pDelta_NetO, d_pTempPD_NetOToNetH, d_pDelta_NetH, d_pLabel60000, d_pLabel);
	}
	/******************** 将结果复制回主机 ********************/ 
	HANDLE_ERROR( hipMemcpy(pNetInput60000, d_pNetInput60000, sizeof(float)*48287718, hipMemcpyDeviceToHost) );
}

/******************** 读【训练集】图片文件 ********************/
int ReadTrainPicData(unsigned char*data, unsigned char*pLabel60000)
{
	int iCheck = 0;

	FILE *fp_image = fopen("MNIST/train-images.idx", "rb");//打开这个二进制(b)文件train-images.idx，以只读(r)的方式打开，
	FILE *fp_label = fopen("MNIST/train-labels.idx", "rb");//读类标
	//FILE *fp_image = fopen("MNIST/t10k_images.idx", "rb");//打开这个二进制(b)文件t10k-images.idx，以只读(r)的方式打开，
	//FILE *fp_label = fopen("MNIST/t10k_labels.idx", "rb");//读类标
	if (!fp_image || !fp_label) 
		return 1;

	fseek(fp_image, 16, SEEK_SET);//从文件开始位置偏移16字节，定位像素数据开始
	iCheck = fread(data, sizeof(char)*pixels*pixels, TestNumber, fp_image);//从文件(fp_image)中读(pixels * pixels)字节数，读TestNumber次，到data指向的内存中
	if (iCheck != TestNumber)
	{
		printf("读图片文件失败！\n");
	}
	else
	{
		printf("读图片文件成功！\n");	
	}

	fseek(fp_label, 8, SEEK_SET);//从文件开始位置偏移8字节，定位标签数据
	iCheck = fread(pLabel60000, sizeof(char), TestNumber, fp_label);//从文件(fp_label)中读sizeof(char)个字节数，读TestNumber次，读到pLabel指向的内存中
	if (iCheck != TestNumber)
	{
		printf("读Label文件失败！\n");
	}
	else
	{
		printf("读Label文件成功！\n");	
	}

	fclose(fp_image);
	fclose(fp_label);

	return 0;
}

/******************** 读【测试集】图片文件 ********************/
int ReadTest10kPicData(unsigned char*data, unsigned char*pLabel60000)
{
	int iCheck = 0;

	FILE *fp_image = fopen("MNIST/t10k_images.idx", "rb");//打开这个二进制(b)文件t10k-images.idx，以只读(r)的方式打开，
	FILE *fp_label = fopen("MNIST/t10k_labels.idx", "rb");//读类标
	if (!fp_image || !fp_label) 
		return 1;

	fseek(fp_image, 16, SEEK_SET);//从文件开始位置偏移16字节，定位像素数据开始
	iCheck = fread(data, sizeof(char)*pixels*pixels, CheckNumber, fp_image);//从文件(fp_image)中读(pixels * pixels)字节数，读CheckNumber次，到data指向的内存中
	if (iCheck != CheckNumber)
	{
		printf("读图片文件失败！\n");
	}
	else
	{
		printf("读图片文件成功！\n");	
	}

	fseek(fp_label, 8, SEEK_SET);//从文件开始位置偏移8字节，定位标签数据
	iCheck = fread(pLabel60000, sizeof(char), CheckNumber, fp_label);//从文件(fp_label)中读sizeof(char)个字节数，读CheckNumber次，读到pLabel指向的内存中
	if (iCheck != CheckNumber)
	{
		printf("读Label文件失败！\n");
	}
	else
	{
		printf("读Label文件成功！\n");	
	}

	fclose(fp_image);
	fclose(fp_label);

	return 0;
}


/******************** 随机初始化权重 ********************/
void RandomInitWeight(float *WeightIToH, float *WeightHToO)
{
	for(int m = 0; m < N*N; m++)
	{
		*(WeightIToH + m) = (rand()/(float)(RAND_MAX))/2000; //【输入层-隐藏层】之间任意2个神经元间的权重W
	}
	printf("所有输入层-隐藏层间的权重数据已经初始化完毕！\n");

	for(int n = 0; n < N*M; n++)
	{
		*(WeightHToO + n) = (rand()/(float)(RAND_MAX))/400; //【隐藏层-输出层】之间任意2个神经元间的权重W
	}
	printf("所有隐藏层-输出层间的权重数据已经初始化完毕！\n");
}

/******************** 写文件【训好的权重】 ********************/
void WriteWeightData(float *WeightIToH, float *WeightHToO)
{
	int iNum = 0;
	FILE *fpWriteInputToHidden = fopen("MNIST02/fWeightItoHData201810161953.wet","wb+");
	FILE *fpWriteHiddenToOutput = fopen("MNIST02/fWeightHtoOData201810161953.wet","wb+");
	if(fpWriteInputToHidden == NULL)
	{
		printf("创建fWeightItoHData201810161953.wet文件失败！\n");
		return;
	}
	if(fpWriteHiddenToOutput == NULL)
	{
		printf("创建fWeightHtoOData201810161953.wet文件失败！\n");
		return;
	}

	//将输入层-隐藏层间的权重写入到WeightItoHData.wet文件中
	iNum = fwrite(WeightIToH, sizeof(float), N*N, fpWriteInputToHidden);
	if (iNum != N*N)
	{
		printf("写fWeightItoHData201810161953.wet文件失败！");
		return;
	}
	printf("所有输入层-隐藏层间的权重数据已经保存\n");


	//将隐藏层-输出层间的权重写入到WeightHtoOData.wet文件中
	iNum = fwrite(WeightHToO, sizeof(float), N*M, fpWriteHiddenToOutput);
	if (iNum != N*M)
	{		
		printf("写fWeightHtoOData201810161953.wet文件失败！");
		return;
	}
	printf("所有隐藏层-输出层间的权重数据已经保存\n");

	fclose(fpWriteInputToHidden);
	fclose(fpWriteHiddenToOutput);
	return;
}

/******************** 读文件【权重WeightInputToHidden】 ********************/
void ReadWeightInputToHiddenData(float *WeightIToH)
{
	int iNum = 0;//输入层-隐藏层间的神经元个数

	FILE *fpReadWeightIToH = fopen("MNIST01/fWeightItoHData201810161953.wet","rb");
	if(fpReadWeightIToH == NULL)
	{
		printf("fWeightItoHData201810161953.wet文件不存在！\n");
		return;
	}

	//下面是读数据，将读到的数据存到数组WeightInputToHidden[NeuronNumHidden]中
	iNum = fread(WeightIToH, sizeof(float), N*N, fpReadWeightIToH);
	if(iNum != N*N)
	{
		printf("读数据存到WeightIToH指向的内存错误！！！\n");
		return;
	}
	printf("读【输入层-隐藏层】间的权重数据成功\n");
	fclose(fpReadWeightIToH);
	return;
}

/******************** 读文件【权重WeightHiddenToOutput】 ********************/
void ReadWeightHiddenToOutputData(float *WeightHToO)
{
	int iNum = 0;//隐藏层-输出层间的神经元个数

	FILE *fpReadWeightHToO = fopen("MNIST01/fWeightHtoOData201810161953.wet","rb");
	if(fpReadWeightHToO == NULL)
	{
		printf("fWeightHtoOData201810161953.wet文件不存在！\n");
		return;
	}

	//下面是读数据，将读到的数据存到数组WeightHiddenToOutput[]中
	iNum = fread(WeightHToO, sizeof(float), N*M, fpReadWeightHToO);
	if(iNum != N*M)
	{
		printf("读数据存到WeightHToO指向的内存错误！！！\n");
	}
	printf("读【隐藏层-输出层】间的权重数据成功\n");
	fclose(fpReadWeightHToO);
	return;
}

/******************** 强制类型转换 ********************/
void TypeCasting()
{
	for(int iNum = 0; iNum < N*TestNumber; iNum++)
	{
		*(pNetInput60000 + iNum) = (float)(*(pPicture + iNum));
	}
}

