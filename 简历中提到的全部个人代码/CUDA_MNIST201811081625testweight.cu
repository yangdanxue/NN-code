/****************************************************************************************************************
父文件：/home/yangdanxue/CUDA_MNIST201811081625（202010202200，【CNN-784-784-196-10@1】将扩充图片pPadding_NetInput移至GPU完成。已成功跑通，且正确率能在震荡中提升）.cu

参考文件：/home/yangdanxue/201811081625.c
		/home/yangdanxue/CUDA_MNIST201810161953（202008292315，优化代码***** KEY *****一次上传一张图片到GPU，1个global，随机权重。pPD_WeightHiddenToOutput层改为784个线程并行后，成功跑通啦。训1轮用时4'25''~~~!!!）.cu
		/home/yangdanxue/CUDA_MNIST201811081625（202010132255，【CNN-784-784-196-10@1】截至到所有权重均已更新完毕。13s左右60000张图训练1轮。问题：正确率在80%附近震荡。pPD_WeightInputToConvolution1值和手动计算不一致）.cu

当前版本进度： 20220212更新：读取【正确率：90.6467%】测试测试集：【正确数：9097 正确率：90.9700%】。
*****************************************************************************************************************/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define Padding				1024
#define N					784
#define M					10
#define K					25
#define S					196



#define pixels				28		//图片像素是28*28
#define Padding_pixels			32		//填充后图片像素是32*32
#define ConvolutionKernel		5		//卷积核，大小是5*5
#define NeuronNumConvolution1		28		//卷积1层神经元个数
#define NeuronNumSubsampling2		14		//池化2层神经元个数
#define Filter				2		//池化层过滤器大小
#define NeuronNumOutput			10		//输出层神经元个数

#define TestNumber			10000		//总测试图片的数量

#define OldWeightPercentage		0.9   	//反向传播更新权重时旧权重占比
#define NewWeightPercentage		0.1   	//反向传播更新权重时新权重占比
#define LearningRate			0.01		//学习速率

int ReadPicData(unsigned char*, unsigned char*); 	//读图片数据文件

void WriteWeightData(float*, float*);			//保留当前调好的权重
void ReadWeightData(float*, float*);			//使用保存的权重
void InitializationWeight(float*, float*);		//随机初始化各值

unsigned char iPicture[pixels][pixels] = {0};		//导入的MNIST数据库图片数组
unsigned char *pLabel60000;					//指向读取的60000张标签数组的一维指针
unsigned char *pLabel;					//用于反向传播的指向60000张标签的指针，用于当前具体指向哪张图片+标签进行测试+训练（因为是用于定位的指针，不是为了指向开了空间的指针，所以无需开空间+释放空间）

float *pNetInput60000;							 	 	 		//指向读取的60000张图片数组的一维指针
float *pNetInput,						*d_pNetInput;					//指向每次只上传1张图片数据至GPU的指针，的主机副本，与，设备副本
float *pPadding_NetInput,					*d_pPadding_NetInput;			//对于5*5的过滤器，在导入的MNIST数据库图片数组iPicture[y][x]周围，填充宽度为2的0时，就可以保证输出图像和输入图像一样大

//各层权重
float *pWeightInputToConvolution1,				*d_pWeightInputToConvolution1;		//【输入层-卷积1层】卷积核（权重）
float *pWeightSubsampling2ToOutput,			*d_pWeightSubsampling2ToOutput;		//【池化2层-输出层】权重

//各层神经元值 & 激活函数值（Sigmoid函数）
float *pNetConvolution1,					*d_pNetConvolution1;				//卷积1层神经元的值
float *pSigmoidConvolution1,				*d_pSigmoidConvolution1;			//卷积1层激活函数值（Sigmoid函数）

//设定最大池化值的坐标结构体数组
struct MAXPool 
{
	float Value;
	float x;
	float y;
	float i;
	float j;
};struct MAXPool *pNetSubsampling2, *d_pNetSubsampling2;							//池化2层神经元的值

float *pSigmoidSubsampling2,				*d_pSigmoidSubsampling2;			//池化2层激活函数值（Sigmoid函数）
float *pNetOutput,						*d_pNetOutput;				//输出层神经元的值
float *pSigmoidOutput,					*d_pSigmoidOutput;				//输出层激活函数值（Sigmoid函数）

//目标输出值
float *pTarget,						*d_pTarget;					//Target的主机副本，与，设备副本	

//输出层---->池化2层的权值更新
float *pPD_ErrorToSigmoidOutput,				*d_pPD_ErrorToSigmoidOutput;
float *pPD_SigmoidOutputToNetOutput,			*d_pPD_SigmoidOutputToNetOutput;
float *pDelta_NetOutput,					*d_pDelta_NetOutput;
float *pPD_NetOutputToWeightSubsampling2ToOutput,	*d_pPD_NetOutputToWeightSubsampling2ToOutput;
float *pPD_WeightSubsampling2ToOutput,			*d_pPD_WeightSubsampling2ToOutput;

//池化2层---->卷积1层的权值更新
float *pPD_NetOutputToSigmoidSubsampling2,		*d_pPD_NetOutputToSigmoidSubsampling2;
float *pPD_SigmoidSubsampling2ToNetSubsampling2,		*d_pPD_SigmoidSubsampling2ToNetSubsampling2;
float *pDelta_Subsampling2,					*d_pDelta_Subsampling2;
float *pUpsample_Delta_Subsampling2,			*d_pUpsample_Delta_Subsampling2;
float *pPD_SigmoidConvolution1ToNetConvolution1,		*d_pPD_SigmoidConvolution1ToNetConvolution1;
float *pDelta_Convolution1,					*d_pDelta_Convolution1;
float *pPD_WeightInputToConvolution1,			*d_pPD_WeightInputToConvolution1;

int Correct = 0;//识别正确数
//***********************************************************************************************************

////////////////////////////////////////////	前向传播（开始）	////////////////////////////////////////////
//计算sigmoid的值
__device__ __forceinline__ float sigmoid (float a)
{
    return 1.0 / (1.0 + exp (-a));
}

__global__ void ForwardAndBackPropagation(float *pNetInput, float *pPadding_NetInput, float *pWeightInputToConvolution1, float *pNetConvolution1, float *pSigmoidConvolution1, struct MAXPool *pNetSubsampling2, float *pSigmoidSubsampling2, float *pWeightSubsampling2ToOutput, float *pNetOutput, float *pSigmoidOutput, float *pTarget, float *pPD_ErrorToSigmoidOutput, float *pPD_SigmoidOutputToNetOutput, float *pDelta_NetOutput, float *pPD_NetOutputToWeightSubsampling2ToOutput, float *pPD_WeightSubsampling2ToOutput, float *pPD_NetOutputToSigmoidSubsampling2, float *pPD_SigmoidSubsampling2ToNetSubsampling2, float *pDelta_Subsampling2, float *pUpsample_Delta_Subsampling2, float *pPD_SigmoidConvolution1ToNetConvolution1, float *pDelta_Convolution1, float *pPD_WeightInputToConvolution1)
{
	int Y = blockIdx.y*blockDim.y+threadIdx.y;
 	int X = blockIdx.x*blockDim.x+threadIdx.x;

	//在iPicture[y][x]周围，填充2圈0——因为在图像边缘的像素在输出中采用较少，为了不丢掉了很多图像边缘的信息，在图像卷积操作之前，沿着图像边缘用0进行图像填充。对于5*5的过滤器，填充宽度为2时，就可以保证输出图像和输入图像一样大。
    	if (X < 28 && Y < 28) 
	{
		pPadding_NetInput[(Y+2)*32+(X+2)] = pNetInput[Y*28+X];
	}
     	__syncthreads();


	float tmpSum = 0;
   	///////////////////////////【前向传播】计算卷积1层NetConvolution1神经元的值///////////////////////////
    	if (X < 28 && Y < 28) 
	{
     		for (int j = 0; j < 5; j++) 
		{
	     		for (int i = 0; i < 5; i++) 
			{
		   		tmpSum += pPadding_NetInput[(Y+j)*32+(X+i)] * pWeightInputToConvolution1[j*5+i];
			}
		} 
     		pNetConvolution1[Y*28+X] = tmpSum;
		
	   	///////////////////////////【前向传播】计算卷积1层NetConvolution1的sigmoid函数值///////////////////////////
	    	pSigmoidConvolution1[Y*28+X] = sigmoid (pNetConvolution1[Y*28+X]);
     	}
     	__syncthreads();


	///////////////////////////【前向传播】计算池化2层NetSubsampling2神经元的值///////////////////////////
    	if (X < 14 && Y < 14) 
	{
		pNetSubsampling2[Y*14+X].Value = 0;
		pNetSubsampling2[Y*14+X].i = 0;
		pNetSubsampling2[Y*14+X].j = 0;

		for(int j = 0; j < 2; j++)
		{
			for (int i = 0; i < 2; i++)
			{
				//求出每2*2过滤器中的最大值，组成池化层
				if (pSigmoidConvolution1[(Y*2+j)*28+(X*2+i)] > pNetSubsampling2[Y*14+X].Value)
				{
					pNetSubsampling2[Y*14+X].Value = pSigmoidConvolution1[(Y*2+j)*28+(X*2+i)];
					pNetSubsampling2[Y*14+X].i = i;
					pNetSubsampling2[Y*14+X].j = j;
				}
			}
		}

	   	///////////////////////////【前向传播】计算池化2层NetSubsampling2的sigmoid函数值///////////////////////////
//	    	pSigmoidSubsampling2[Y*14+X] = pNetSubsampling2[Y*14+X].j;		//验证i、j值都正确啦（与【pNetSubsampling2[Y*14+X].i = i】与【pNetSubsampling2[Y*14+X].j = j】两行代码匹配）。
	    	pSigmoidSubsampling2[Y*14+X] = pNetSubsampling2[Y*14+X].Value;	//全部写成数组形式，pSigmoidSubsampling2值全部正确
	}
     	__syncthreads();


	////////////////【前向传播】计算输出层NetOutput神经元的值////////////////
	tmpSum = 0;

	if (X < M && Y == 0) 
	{
		for (int i = 0; i < 196; i++)
		{
           		tmpSum += pWeightSubsampling2ToOutput[X * 196 + i] * pSigmoidSubsampling2[i];
		}
    		pNetOutput[X] = tmpSum;
    		
    		///////////////////////////【前向传播】计算输出层NetOutput的sigmoid函数值///////////////////////////
    		pSigmoidOutput[X] = sigmoid (pNetOutput[X]);
    	}
	__syncthreads();
}



//*****************************************	以上为GPU上运行的代码	*****************************************//

int main(void) 
{
	unsigned char Size = sizeof(float);
	unsigned char *pPicture;
	

	// a、b、c设备副本的分配空间
	hipMalloc((void **)&d_pWeightInputToConvolution1, Size*K);
	hipMalloc((void **)&d_pWeightSubsampling2ToOutput, Size*(S*M));
	hipMalloc((void **)&d_pNetInput, Size*N);
	hipMalloc((void **)&d_pPadding_NetInput, Size*Padding);
	hipMalloc((void **)&d_pNetConvolution1, Size*N);
	hipMalloc((void **)&d_pSigmoidConvolution1, Size*N);
	hipMalloc((void **)&d_pNetSubsampling2, Size*S*5);
	hipMalloc((void **)&d_pSigmoidSubsampling2, Size*S);
	hipMalloc((void **)&d_pNetOutput, Size*M);
	hipMalloc((void **)&d_pSigmoidOutput, Size*M);
	////////////////////////////////////////////////////////////////////////
	hipMalloc((void **)&d_pTarget, Size*M);
	hipMalloc((void **)&d_pPD_ErrorToSigmoidOutput, Size*M);
	hipMalloc((void **)&d_pPD_SigmoidOutputToNetOutput, Size*M);	
	hipMalloc((void **)&d_pDelta_NetOutput, Size*M);					
	hipMalloc((void **)&d_pPD_NetOutputToWeightSubsampling2ToOutput, Size*(S*M));
	hipMalloc((void **)&d_pPD_WeightSubsampling2ToOutput, Size*(S*M));
	////////////////////////////////////////////////////////////////////////	
	hipMalloc((void **)&d_pPD_NetOutputToSigmoidSubsampling2, Size*(S*M));
	hipMalloc((void **)&d_pPD_SigmoidSubsampling2ToNetSubsampling2, Size*S);
	hipMalloc((void **)&d_pDelta_Subsampling2, Size*S);
	hipMalloc((void **)&d_pUpsample_Delta_Subsampling2, Size*N);
	hipMalloc((void **)&d_pPD_SigmoidConvolution1ToNetConvolution1, Size*N);	
	hipMalloc((void **)&d_pDelta_Convolution1, Size*N);		
	////////////////////////////////////////////////////////////////////////	
	hipMalloc((void **)&d_pPD_WeightInputToConvolution1, Size*K);		


	//为a、b、c和设置输入值的主机副本分配空间
	pWeightInputToConvolution1 = (float*)malloc(Size*K);
	pWeightSubsampling2ToOutput = (float*)malloc(Size*(S*M));
	pPicture = (unsigned char*)malloc(N*TestNumber);
	pLabel60000 = (unsigned char*)malloc(TestNumber);
	pNetInput60000 = (float*)malloc(Size*(N*TestNumber));
	pPadding_NetInput = (float*)malloc(Size*Padding);
	pNetConvolution1 = (float*)malloc(Size*N);
	pSigmoidConvolution1 = (float*)malloc(Size*N);
	pNetSubsampling2 = (struct MAXPool*)malloc(Size*S*5);
	pSigmoidSubsampling2 = (float*)malloc(Size*S);
	pNetOutput = (float*)malloc(Size*M);	
	pSigmoidOutput = (float*)malloc(Size*M);
	////////////////////////////////////////////////////////////////////////
	pTarget = (float*)malloc(Size*M);			
	pPD_ErrorToSigmoidOutput = (float*)malloc(Size*M);	
	pPD_SigmoidOutputToNetOutput = (float*)malloc(Size*M);
	pDelta_NetOutput = (float*)malloc(Size*M);	
	pPD_NetOutputToWeightSubsampling2ToOutput = (float*)malloc(Size*(S*M));
	pPD_WeightSubsampling2ToOutput = (float*)malloc(Size*(S*M));
	////////////////////////////////////////////////////////////////////////	
	pPD_NetOutputToSigmoidSubsampling2 = (float*)malloc(Size*(S*M));
	pPD_SigmoidSubsampling2ToNetSubsampling2 = (float*)malloc(Size*S);
	pDelta_Subsampling2 = (float*)malloc(Size*S);
	pUpsample_Delta_Subsampling2 = (float*)malloc(Size*N);		
	pPD_SigmoidConvolution1ToNetConvolution1 = (float*)malloc(Size*N);
	pDelta_Convolution1 = (float*)malloc(Size*N);				
	////////////////////////////////////////////////////////////////////////		
	pPD_WeightInputToConvolution1 = (float*)malloc(Size*K);	
	
			
	
	//读权重
	ReadWeightData(pWeightInputToConvolution1, pWeightSubsampling2ToOutput);
//	InitializationWeight(pWeightInputToConvolution1, pWeightSubsampling2ToOutput);

	//将输入复制到设备
	hipMemcpy(d_pWeightInputToConvolution1,  pWeightInputToConvolution1,  Size*K, hipMemcpyHostToDevice);
	hipMemcpy(d_pWeightSubsampling2ToOutput, pWeightSubsampling2ToOutput, Size*(S*M), hipMemcpyHostToDevice);
			
	//读图片和标签
	ReadPicData(pPicture, pLabel60000);

	for (int k = 1; k < 2; k++)
	{
		int result = 0;	//声明在main()开始处，会导致pSigmoidOutput[0]不参与if判断，从而所有的图片0都会被识别为除pSigmoidOutput[0]值外的最大值对应的元素数字。由i=1，改为i=0即可。
		int Correct = 0;	//识别正确数
		float CorrectRate = 0;


		printf("第%d次训练：\n", k);
		
		for(int i = 0; i < N*TestNumber; i++)
		{
			*(pNetInput60000 + i) = (float)(*(pPicture + i));	//新开一个指针pNetInput60000，指向一次性读取60000张的图片数组。该指针始终指向60000张的图片数组的首地址。
		}


		pLabel = pLabel60000;	//pLabel：用于反向传播的指向60000张标签的指针，用于当前具体指向哪张图片+标签进行测试+训练（因为是用于定位的指针，不是为了指向开了空间的指针，所以无需开空间+释放空间）。
						//在未声明指针位置时，指针默认指向内存0地址。【pLabel = pLabel60000;】此赋值语句表示，让pLabelBP60000指向与pLabel60000相同的地址。

		for(int iNum = 0; iNum < TestNumber; iNum++)
		{
			pNetInput = pNetInput60000 + iNum*N;	

			for (int i = 0; i < 10; i++)
			{
				*(pTarget + i) = 0;	//清理数组Target[]
			}

			pLabel = pLabel60000 + iNum;
			*(pTarget + *pLabel) = 1;	//语句想要实现的目的：给数组Target的第label个元素赋值为1，之前的写法是【*(pTarget + pLabel60000) = 1】
								//【pLabel】是反向传播中用于指向60000张标签的定位用的指针，【*pLabel】是该指针指向的地址中存放的值。
								//比如【pLabel】指向第一张图片5的地址，【*pLabel】=5。【&pLabel】的意思是存放指针pLabelBP60000变量的地址。
			//将输入复制到设备
			hipMemcpy(d_pNetInput,			pNetInput,			Size*N,		hipMemcpyHostToDevice);
			hipMemcpy(d_pTarget,			pTarget,			Size*M,		hipMemcpyHostToDevice);
			
			dim3 blocksPerGrid(1, 1);
			dim3 threadsPerBlock(28, 28);
			
			//在带有N个块的GPU上启动add（）内核
			ForwardAndBackPropagation<<<blocksPerGrid, threadsPerBlock>>>(d_pNetInput, d_pPadding_NetInput, d_pWeightInputToConvolution1, d_pNetConvolution1, d_pSigmoidConvolution1, d_pNetSubsampling2, d_pSigmoidSubsampling2, d_pWeightSubsampling2ToOutput, d_pNetOutput, d_pSigmoidOutput, d_pTarget, d_pPD_ErrorToSigmoidOutput, d_pPD_SigmoidOutputToNetOutput, d_pDelta_NetOutput, d_pPD_NetOutputToWeightSubsampling2ToOutput, d_pPD_WeightSubsampling2ToOutput, d_pPD_NetOutputToSigmoidSubsampling2, d_pPD_SigmoidSubsampling2ToNetSubsampling2, d_pDelta_Subsampling2, d_pUpsample_Delta_Subsampling2, d_pPD_SigmoidConvolution1ToNetConvolution1, d_pDelta_Convolution1, d_pPD_WeightInputToConvolution1);


			//将结果复制回主机
//			cudaMemcpy(pNetInput,						d_pNetInput,						Size*N,		cudaMemcpyDeviceToHost);
			hipMemcpy(pPadding_NetInput,					d_pPadding_NetInput,					Size*Padding,	hipMemcpyDeviceToHost);
			hipMemcpy(pNetConvolution1, 		  	 		d_pNetConvolution1, 		   			Size*N, 		hipMemcpyDeviceToHost);
			hipMemcpy(pSigmoidConvolution1, 	   			d_pSigmoidConvolution1, 	  	 			Size*N, 		hipMemcpyDeviceToHost);
			hipMemcpy(pNetSubsampling2, 		  	 		d_pNetSubsampling2, 		   			Size*S*5, 		hipMemcpyDeviceToHost);
			hipMemcpy(pSigmoidSubsampling2, 	   			d_pSigmoidSubsampling2, 	  	 			Size*S, 		hipMemcpyDeviceToHost);
			hipMemcpy(pNetOutput, 		   				d_pNetOutput, 		   				Size*M, 		hipMemcpyDeviceToHost);	
			hipMemcpy(pSigmoidOutput, 	   				d_pSigmoidOutput, 	   				Size*M, 		hipMemcpyDeviceToHost);
			hipMemcpy(pPD_ErrorToSigmoidOutput,				d_pPD_ErrorToSigmoidOutput,				Size*M,		hipMemcpyDeviceToHost);
			hipMemcpy(pPD_SigmoidOutputToNetOutput,			d_pPD_SigmoidOutputToNetOutput,			Size*M,		hipMemcpyDeviceToHost);
			hipMemcpy(pDelta_NetOutput,					d_pDelta_NetOutput,					Size*M,		hipMemcpyDeviceToHost);
			hipMemcpy(pPD_NetOutputToWeightSubsampling2ToOutput,	d_pPD_NetOutputToWeightSubsampling2ToOutput,	Size*(S*M),		hipMemcpyDeviceToHost);
			hipMemcpy(pPD_WeightSubsampling2ToOutput, 		d_pPD_WeightSubsampling2ToOutput, 			Size*(S*M),		hipMemcpyDeviceToHost);
			hipMemcpy(pWeightSubsampling2ToOutput, 			d_pWeightSubsampling2ToOutput, 			Size*(S*M),		hipMemcpyDeviceToHost);
			hipMemcpy(pPD_NetOutputToSigmoidSubsampling2, 		d_pPD_NetOutputToSigmoidSubsampling2, 		Size*(S*M),		hipMemcpyDeviceToHost);
			hipMemcpy(pPD_SigmoidSubsampling2ToNetSubsampling2, 	d_pPD_SigmoidSubsampling2ToNetSubsampling2, 	Size*S,		hipMemcpyDeviceToHost);
			hipMemcpy(pDelta_Subsampling2, 				d_pDelta_Subsampling2, 					Size*S,		hipMemcpyDeviceToHost);				
			hipMemcpy(pUpsample_Delta_Subsampling2, 			d_pUpsample_Delta_Subsampling2, 			Size*N,		hipMemcpyDeviceToHost);
			hipMemcpy(pPD_SigmoidConvolution1ToNetConvolution1, 	d_pPD_SigmoidConvolution1ToNetConvolution1,	Size*N,		hipMemcpyDeviceToHost);
			hipMemcpy(pDelta_Convolution1, 				d_pDelta_Convolution1,					Size*N,		hipMemcpyDeviceToHost);
			hipMemcpy(pPD_WeightInputToConvolution1, 		  	d_pPD_WeightInputToConvolution1, 		   	Size*K, 		hipMemcpyDeviceToHost);
			hipMemcpy(pWeightInputToConvolution1, 			d_pWeightInputToConvolution1, 				Size*K,		hipMemcpyDeviceToHost);



			for (int i = 0; i < M; i++)	//声明在main()开始处，会导致pSigmoidOutput[0]不参与if判断，从而所有的图片0都会被识别为除pSigmoidOutput[0]值外的最大值对应的元素数字。由i=1，改为i=0即可（20200705改）。
			{
				if (pSigmoidOutput[result] < pSigmoidOutput[i]) //10个值比大小，哪个大说明网络把图片识别为几
				{
					result = i;
				}	
			}
/*			printf("读取第%d张图片是%d \n",iNum+1, *pLabel);
			printf("被网络识别为[%d] \n", result);
			
			for(int i = 0; i < 10; i++)
			{
				printf("pSigmoidOutput[%d]=%e\n", i, *(pSigmoidOutput + i));
			}
*/	
			
			if (result == *pLabel)
			{	
				Correct++;
			}
		}	
		CorrectRate = ((float)Correct) / TestNumber;
		printf("正确数：%d 正确率：%.4f%%\n", Correct, CorrectRate * 100);
	}

}








//************************************************************************************************************************
int ReadPicData(unsigned char*data, unsigned char*pLabel60000)
{
	int iCheck = 0;

//	FILE *fp_image = fopen("MNIST/train-images.idx", "rb");//打开这个二进制(b)文件train-images.idx，以只读(r)的方式打开，
//	FILE *fp_label = fopen("MNIST/train-labels.idx", "rb");//读类标
	FILE *fp_image = fopen("MNIST/t10k_images.idx", "rb");//打开这个二进制(b)文件t10k-images.idx，以只读(r)的方式打开，
	FILE *fp_label = fopen("MNIST/t10k_labels.idx", "rb");//读类标
	if (!fp_image || !fp_label) 
		return 1;

	fseek(fp_image, 16, SEEK_SET);//从文件开始位置偏移16字节，定位像素数据开始
	iCheck = fread(data, sizeof(char)*pixels*pixels, TestNumber, fp_image);//从文件(fp_image)中读(pixels * pixels)字节数，读TestNumber次，到data指向的内存中
	if (iCheck != TestNumber)
	{
		printf("读图片文件失败！\n");
	}
	else
	{
		printf("读图片文件成功！\n");	
	}

	fseek(fp_label, 8, SEEK_SET);//从文件开始位置偏移8字节，定位标签数据
	iCheck = fread(pLabel60000, sizeof(char), TestNumber, fp_label);//从文件(fp_label)中读sizeof(char)个字节数，读TestNumber次，读到pLabel指向的内存中
	if (iCheck != TestNumber)
	{
		printf("读Label文件失败！\n");
	}
	else
	{
		printf("读Label文件成功！\n");	
	}

	fclose(fp_image);
	fclose(fp_label);

	return 0;
}


//读权重——使用保存的权重
void ReadWeightData(float *WeightInputToConvolution1, float *WeightSubsampling2ToOutput)
{
	int iNum = 0; //神经元个数

	FILE *fpReadItoC1 = fopen("MNIST01/fWeightItoC1Data201811081625.wet","rb");
	FILE *fpReadS2toO = fopen("MNIST01/fWeightS2toOData201811081625.wet","rb");
	
	if(fpReadItoC1 == NULL)
	{
		printf("fWeightItoC1Data201811081625.wet文件不存在！\n");
		return;
	}
	if(fpReadS2toO == NULL)
	{
		printf("fWeightS2toOData201811081625.wet文件不存在！\n");
		return;
	}

	//下面是读数据，将读到的数据存到数组WeightInputToConvolution1[NeuronNumConvolution1]中
	iNum = fread(WeightInputToConvolution1, sizeof(float), ConvolutionKernel * ConvolutionKernel, fpReadItoC1);
	if(iNum != ConvolutionKernel * ConvolutionKernel)
	{
		printf("读数据存到数组WeightInputToConvolution1[ConvolutionKernel * ConvolutionKernel]错误！！！\n");
	}
	printf("读[输入层-卷积1层]间的权重数据成功\n");

	//下面是读数据，将读到的数据存到数组WeightSubsampling2ToOutput[NeuronNumSubsampling2 * NeuronNumOutput]中
	iNum = fread(WeightSubsampling2ToOutput, sizeof(float), NeuronNumSubsampling2 * NeuronNumSubsampling2 * NeuronNumOutput, fpReadS2toO);
	if(iNum != NeuronNumSubsampling2 * NeuronNumSubsampling2 * NeuronNumOutput)
	{
		printf("读数据存到数组WeightSubsampling2ToOutput[NeuronNumSubsampling2 * NeuronNumSubsampling2 * NeuronNumOutput]错误！！！\n");
	}
	printf("读[池化2层-输出层]间的权重数据成功\n");

	fclose(fpReadItoC1);
	fclose(fpReadS2toO);

	return;
}



//写权重——保留当前训好的权重
void WriteWeightData(float *WeightInputToConvolution1, float *WeightSubsampling2ToOutput)
{
	int iNum = 0;
	FILE *fpWriteInputToConvolution1 = fopen("MNIST02/fWeightItoC1Data201811081625.wet","wb+");
	FILE *fpWriteSubsampling2ToOutput = fopen("MNIST02/fWeightS2toOData201811081625.wet","wb+");

	if(fpWriteInputToConvolution1 == NULL)
	{
		printf("创建WeightItoC1Data201811081625.wet文件失败！\n");
		return;
	}
	if(fpWriteSubsampling2ToOutput == NULL)
	{
		printf("创建WeightS2toOData201811081625.wet文件失败！\n");
		return;
	}

	//将输入层-卷积1层间的权重写入到WeightItoC1Data201811081625.wet文件中
	iNum = fwrite(WeightInputToConvolution1, sizeof(float), ConvolutionKernel * ConvolutionKernel, fpWriteInputToConvolution1);
	if (iNum != ConvolutionKernel * ConvolutionKernel)
	{
		printf("写WeightItoC1Data201811081625.wet文件失败！");
	}
	printf("所有输入层-卷积1层间的权重数据已经保存\n");

	//将池化2层-输出层间的权重写入到WeightS2toOData201811081625.wet文件中
	iNum = fwrite(WeightSubsampling2ToOutput, sizeof(float), NeuronNumSubsampling2 * NeuronNumSubsampling2 * NeuronNumOutput, fpWriteSubsampling2ToOutput);
	if (iNum != NeuronNumSubsampling2 * NeuronNumSubsampling2 * NeuronNumOutput)
	{		
		printf("写WeightS2toOData201811081625.wet文件失败！");
	}
	printf("所有池化2层-输出层间的权重数据已经保存\n");

	fclose(fpWriteInputToConvolution1);
	fclose(fpWriteSubsampling2ToOutput);
}



/*
//写权重——保留当前调好的[输入层-卷积1层]权重
void WriteWeightInputToConvolution1Data(float *WeightInputToConvolution1)
{
	int iNum = 0;
	FILE *fpWriteInputToConvolution1 = fopen("MNIST/fWeightItoC1Data201811081625.wet","wb+");

	if(fpWriteInputToConvolution1 == NULL)
	{
		printf("创建WeightItoC1Data201811081625.wet文件失败！\n");
		return;
	}

	//将输入层-卷积1层间的权重写入到WeightItoC1Data201811081625.wet文件中
	iNum = fwrite(WeightInputToConvolution1, sizeof(float), ConvolutionKernel * ConvolutionKernel, fpWriteInputToConvolution1);
	if (iNum != ConvolutionKernel * ConvolutionKernel)
	{
		printf("写WeightItoC1Data201811081625.wet文件失败！");
	}
	printf("所有输入层-卷积1层间的权重数据已经保存\n");

	fclose(fpWriteInputToConvolution1);
}


//写权重——保留当前调好的[池化2层-输出层]权重
void WriteWeightSubsampling2ToOutputData(float *WeightSubsampling2ToOutput)
{
	int iNum = 0;
	FILE *fpWriteSubsampling2ToOutput = fopen("MNIST/fWeightS2toOData201811081625.wet","wb+");

	if(fpWriteSubsampling2ToOutput == NULL)
	{
		printf("创建WeightS2toOData201811081625.wet文件失败！\n");
		return;
	}

	//将池化2层-输出层间的权重写入到WeightS2toOData201811081625.wet文件中
	iNum = fwrite(WeightSubsampling2ToOutput, sizeof(float), NeuronNumSubsampling2 * NeuronNumSubsampling2 * NeuronNumOutput, fpWriteSubsampling2ToOutput);
	if (iNum != NeuronNumSubsampling2 * NeuronNumSubsampling2 * NeuronNumOutput)
	{		
		printf("写WeightS2toOData201811081625.wet文件失败！");
	}
	printf("所有池化2层-输出层间的权重数据已经保存\n");

	fclose(fpWriteSubsampling2ToOutput);
}			


//随机初始化[输入层-卷积1层]权重
void InitializationWeightInputToConvolution1(float *WeightInputToConvolution1)
{
//	int m = 0;
//	int n = 0;

	for(int m = 0; m < ConvolutionKernel * ConvolutionKernel; m++)
	{
		//WeightInputToConvolution1[m] = 0.5/2000; //【输入层-卷积1层】之间任意2个神经元间的权重W
		WeightInputToConvolution1[m] = (rand()/(float)(RAND_MAX))/1000; //【输入层-卷积1层】之间任意2个神经元间的权重W
		//WeightInputToConvolution1[m] = 0; //【输入层-卷积1层】之间任意2个神经元间的权重W
		//WeightInputToConvolution1[m] = -0.008001877380; //89.9467%版权重的中位数
		//WeightInputToConvolution1[m] = 0.078278135; //89.9467%版权重的平均数
	}

	//【输入层-卷积1层】权重翻转180度
	//for(m = 0, n = ConvolutionKernel * ConvolutionKernel - 1; m < ConvolutionKernel * ConvolutionKernel, -1 < n ; m++, n--)
	//{
	//	WeightRot180InputToConvolution1[n] = WeightInputToConvolution1[m];
	//}
}


//随机初始化[池化2层-输出层]权重
void InitializationWeightSubsampling2ToOutput(float *WeightSubsampling2ToOutput)
{
//	int m = 0;

	for(int m = 0; m < NeuronNumSubsampling2 * NeuronNumSubsampling2 * NeuronNumOutput; m++)
	{
		//WeightSubsampling2ToOutput[m] = 0; //【池化2层-输出层】之间任意2个神经元间的权重W
		WeightSubsampling2ToOutput[m] = (rand()/(float)(RAND_MAX))/200; //【池化2层-输出层】之间任意2个神经元间的权重W
		//WeightSubsampling2ToOutput[m] = -0.065319700309; ///89.9467%版权重的中位数
		//WeightSubsampling2ToOutput[m] = -0.060853877; ///89.9467%版权重的平均数
	}
}
*/

//随机初始化权重
void InitializationWeight(float *WeightInputToConvolution1, float *WeightSubsampling2ToOutput)
{
	for(int m = 0; m < ConvolutionKernel * ConvolutionKernel; m++)
	{
		WeightInputToConvolution1[m] = (rand()/(float)(RAND_MAX))/1000; //【输入层-卷积1层】之间任意2个神经元间的权重W
	}

	for(int m = 0; m < NeuronNumSubsampling2 * NeuronNumSubsampling2 * NeuronNumOutput; m++)
	{
		WeightSubsampling2ToOutput[m] = (rand()/(float)(RAND_MAX))/200; //【池化2层-输出层】之间任意2个神经元间的权重W
	}
}

