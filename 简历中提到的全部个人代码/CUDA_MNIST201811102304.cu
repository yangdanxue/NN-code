#include "hip/hip_runtime.h"
/****************************************************************************************************************
父文件：/home/yangdanxue/CUDA_MNIST201811102304（202012132338，【CNN，784-784@100-196-10】多卷积核的反向传播，已全部更新完毕，随机权重，正确率第一次训练就到了78.0667%，之后一直稳步增长）.cu

参考文件：/home/yangdanxue/CUDA_MNIST201811102304（202010302008，【CNN，784-784@100-196-10】一次性改了前向传播部分代码，读取99.5467%版权重，正确率50.3600%）.cu
	  /home/yangdanxue/test_Assignment（202012032255，测试模拟：多个卷积核，pDelta_Subsampling2[196*100] += pPD_NetOutputToSigmoidSubsampling2[1960*100] * pDelta_NetOutput[10]，单核多核分别都跑通啦）.cu
	  /home/yangdanxue/201811081625.c
	  /home/yangdanxue/201811102304.c

当前版本进度：202208162253更新：开一整块内存，1024个线程并行，一次上传60000张图到gpu，每次计算600张。
*****************************************************************************************************************/
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include "book.cuh"

#define N					784
#define M					10

#define pixels				28
#define TestNumber			60000
#define CheckNumber			60000
#define OldWeightPercentage		0.9
#define NewWeightPercentage		0.1  
#define LearningRate			0.03		
#define GPUcalculateNumber		600

dim3 threadsPerBlock(32, 32);


int ReadTrainPicData(unsigned char*, unsigned char*);
int ReadTest10kPicData(unsigned char*, unsigned char*); 	
void ReadWeightData(float*, float*);
void WriteWeightData(float*, float*);
void RandomInitWeight(float*, float*);

void TypeCasting();
void AllocatePointer();
void RunCUDA(int);

void RecognizeResult();
void JudgeRightOrWrong(int, int);

unsigned char *pPicture;

float *pNetInput60000,			*d_pNetInput60000;
float *pNetInput,				*d_pNetInput;
float *pPadding_NetInput,			*d_pPadding_NetInput;
float *pWeightItoC1,				*d_pWeightItoC1;
float *pWeightS2toO,				*d_pWeightS2toO;
float *pNetConvolution1,			*d_pNetConvolution1;	
float *pSigmoidConvolution1,		*d_pSigmoidConvolution1;
struct MAXPool
{
	float Value;
	int i;
	int j;
};struct MAXPool *pNetSubsampling2,		*d_pNetSubsampling2;
float *pNetOutput,				*d_pNetOutput;			
float *pSigmoidOutput60000,			*d_pSigmoidOutput60000;			
float *pSigmoidOutput,			*d_pSigmoidOutput;			
float *pTarget,				*d_pTarget;	
float *pDelta_NetOutput,			*d_pDelta_NetOutput;
float *pDelta_Subsampling2,			*d_pDelta_Subsampling2;
float *pUpsample_Delta_Subsampling2,	*d_pUpsample_Delta_Subsampling2;
float *pDelta_Convolution1,			*d_pDelta_Convolution1;
float *pPD_WeightItoC1,			*d_pPD_WeightItoC1;
unsigned char *pLabel60000,			*d_pLabel60000;
unsigned char *pLabel,			*d_pLabel;

int iPicNum;		//图片序号
int Recognize;	//图片被识别为数字几
int correct = 0;
int MaxCorrect = 0;
float CorrectRate = 0;

//***********************************************************************************************************
__device__ int d_iNum;

__device__ __forceinline__ float sigmoid (float a)
{
    return 1.0 / (1.0 + exp (-a));
}

__global__ void ForwardAndBackPropagation(float *pNetInput60000, float *pNetInput, float *pPadding_NetInput, float *pWeightItoC1, float *pNetConvolution1, float *pSigmoidConvolution1, struct MAXPool *pNetSubsampling2, float *pWeightS2toO, float *pNetOutput, float *pSigmoidOutput60000, float *pSigmoidOutput, float *pTarget, float *pDelta_NetOutput, float *pDelta_Subsampling2, float *pUpsample_Delta_Subsampling2, float *pDelta_Convolution1, float *pPD_WeightItoC1, unsigned char *pLabel60000, unsigned char *pLabel)
{
	//内存中的指针分配
	pPadding_NetInput =			pNetInput60000 + 						N*TestNumber;
	pWeightItoC1 =				pPadding_NetInput + 					1024;
	pWeightS2toO =				pWeightItoC1 + 						25*100;
	pNetConvolution1 =			pWeightS2toO + 						196*M*100;
	pSigmoidConvolution1 =			pNetConvolution1 + 					N*100;
	pNetSubsampling2 =			(struct MAXPool*)(pSigmoidConvolution1 + 		N*100);
	pNetOutput =				(float*)(pNetSubsampling2 + 				196*100);
	pSigmoidOutput60000 =			pNetOutput + 						M;
	pTarget =					pSigmoidOutput60000 + 					M*TestNumber;			
	pDelta_NetOutput =			pTarget + 							M;	
	pDelta_Subsampling2 =			pDelta_NetOutput + 					M;
	pUpsample_Delta_Subsampling2 =		pDelta_Subsampling2 + 					196*100;	
	pDelta_Convolution1 =			pUpsample_Delta_Subsampling2 +				N*100;				
	pPD_WeightItoC1 =				pDelta_Convolution1 + 					N*100;	
	pLabel60000 =				(unsigned char*)(pPD_WeightItoC1 + 			25*100);

	int Y = threadIdx.y;
 	int X = threadIdx.x;

	for(int j = 0; j < GPUcalculateNumber; j++)
	{
		pNetInput  = 	pNetInput60000 +		N*(d_iNum + j);
		pSigmoidOutput = 	pSigmoidOutput60000 +	M*(d_iNum + j);
		pLabel = 		pLabel60000 +		(d_iNum + j);

		//在iPicture[y][x]周围，填充2圈0——因为在图像边缘的像素在输出中采用较少，为了不丢掉了很多图像边缘的信息，在图像卷积操作之前，沿着图像边缘用0进行图像填充。对于5*5的过滤器，填充宽度为2时，就可以保证输出图像和输入图像一样大。
	    	if (X < 32 && Y < 32) 
		{
			pPadding_NetInput[Y*32+X] = 0;
		}
	     	__syncthreads();

	    	if (X < 28 && Y < 28) 
		{
			pPadding_NetInput[(Y+2)*32+(X+2)] = pNetInput[Y*28+X];
		}
	     	__syncthreads();

	   	///////////////////////////【前向传播】计算卷积1层NetConvolution1神经元的值///////////////////////////
	 	float tmpSum = 0;

		for(int NumK = 0; NumK < 100; NumK++)
		{
		     	tmpSum = 0;

		   	if (X < 28 && Y < 28) 
			{
		     		for (int j = 0; j < 5; j++) 
				{
			     		for (int i = 0; i < 5; i++) 
					{
				   		tmpSum += pPadding_NetInput[(Y+j)*32+(X+i)] * pWeightItoC1[NumK*5*5+(j*5+i)];
					}
				} 
		     		pNetConvolution1[NumK*28*28+(Y*28+X)] = tmpSum;
			    	pSigmoidConvolution1[NumK*28*28+(Y*28+X)] = sigmoid (pNetConvolution1[NumK*28*28+(Y*28+X)]);
		     	}
	 		__syncthreads();
	    	}
		__syncthreads();

	 	///////////////////////////【前向传播】计算池化2层NetSubsampling2神经元的值///////////////////////////
	 	for(int NumK = 0; NumK < 100; NumK++)
		{
		    	if (X < 14 && Y < 14) 
			{
				pNetSubsampling2[NumK*14*14+(Y*14+X)].Value = 0;
				pNetSubsampling2[NumK*14*14+(Y*14+X)].i = 0;
				pNetSubsampling2[NumK*14*14+(Y*14+X)].j = 0;

				for(int j = 0; j < 2; j++)
				{
					for (int i = 0; i < 2; i++)
					{
						//求出每2*2过滤器中的最大值，组成池化层
						if (pSigmoidConvolution1[NumK*28*28+((Y*2+j)*28+(X*2+i))] > pNetSubsampling2[NumK*14*14+(Y*14+X)].Value)	
						{
							pNetSubsampling2[NumK*14*14+(Y*14+X)].Value = pSigmoidConvolution1[NumK*28*28+((Y*2+j)*28+(X*2+i))];
							pNetSubsampling2[NumK*14*14+(Y*14+X)].i = i;
							pNetSubsampling2[NumK*14*14+(Y*14+X)].j = j;
						}
					}
				}
			}
			__syncthreads();
		}
	     	__syncthreads();

		////////////////【前向传播】计算输出层NetOutput神经元的值////////////////
		tmpSum = 0;

		if (X < M && Y == 0) 
		{
			for(int NumK = 0; NumK < 100; NumK++)
			{
				for (int i = 0; i < 196; i++)
				{
			   		tmpSum += pWeightS2toO[X*196*100+NumK*14*14+i] * pNetSubsampling2[NumK*14*14+i].Value;
				}
			}
	    		pNetOutput[X] = tmpSum;
	 	    	pSigmoidOutput[X] = sigmoid (pNetOutput[X]);
	    	}
		__syncthreads();

		if (X < M && Y == 0) 
		{
			pTarget[X] = 0;
			pTarget[*pLabel] = 1;
			pDelta_NetOutput[X] = (-(pTarget[X] - pSigmoidOutput[X])/5) * (pSigmoidOutput[X] * (1 - pSigmoidOutput[X]));
	    	}
		__syncthreads();

		///////////////////////////【反向传播】池化2层---->输出层的权值更新——Error-->pWeightS2toO///////////////////////////	
	     	for (int i = 0; i < M; i++) 
		{
			for(int NumK = 0; NumK < 100; NumK++)   	
			{
				if (X<14 && Y<14) 
				{
					pWeightS2toO[i*196*100 + NumK*196 + Y*14+X] = OldWeightPercentage * (pWeightS2toO[i*196*100 + NumK*196 + Y*14+X]) + NewWeightPercentage * (pWeightS2toO[i*196*100 + NumK*196 + Y*14+X] - LearningRate * (pDelta_NetOutput[i] * pNetSubsampling2[NumK*196 + Y*14+X].Value));			
				}
				__syncthreads();
			}
			__syncthreads();
		}
		__syncthreads();

		///////////////////////////【反向传播】池化2层---->卷积1层的权值更新——Error-->SigmoidSubsampling2(池化2层的误差值＝总误差对池化2层神经元值求偏导)///////////////////////////
		for(int NumK = 0; NumK < 100; NumK++)   	
		{
			tmpSum = 0;

		    	if (X<14 && Y<14) 
			{
			     	for (int i = 0; i < M; i++) 
				{
			   		tmpSum += pWeightS2toO[i*196*100 + NumK*196 + Y*14+X] * pDelta_NetOutput[i];
			   	}
		     	    	pDelta_Subsampling2[NumK*196 + Y*14+X] = tmpSum;
		       	}
			__syncthreads();
		}
		__syncthreads();

		///////////////////////////【反向传播】池化2层---->卷积1层的权值更新——把池化2层的误差值维度还原成上一层（卷积1层）的误差值维度，即除了填充每4格中的最大误差值外，其余位置补0///////////////////////////
		for(int NumK = 0; NumK < 100; NumK++)   	
		{
			if (X < 14 && Y < 14) 
			{
				for(int j = 0; j < 2; j++)
				{
					for (int i = 0; i < 2; i++)
					{
						pUpsample_Delta_Subsampling2[NumK*28*28 + (Y*2+j)*28 + (X*2+i)] = 0;
					}
				}
				pUpsample_Delta_Subsampling2[NumK*28*28 + (Y*2+pNetSubsampling2[NumK*14*14 + Y*14+X].j)*28 + (X*2+pNetSubsampling2[NumK*14*14 + Y*14+X].i)] = pDelta_Subsampling2[NumK*14*14 + Y*14+X];
			}
			__syncthreads();
		}
		__syncthreads();

		///////////////////////////【反向传播】池化2层---->卷积1层的权值更新——卷积1层(池化2层的前一层)的误差值δ＝对池化2层的误差值进行upsample还原 * 卷积1层的激活值对神经元值求偏导///////////////////////////
		for(int NumK = 0; NumK < 100; NumK++)   	
		{
		    	if (X < 28 && Y < 28) 
			{
				pDelta_Convolution1[NumK*28*28 + Y*28+X] = pUpsample_Delta_Subsampling2[NumK*28*28 + Y*28+X] * pSigmoidConvolution1[NumK*28*28 + Y*28+X] * (1 - pSigmoidConvolution1[NumK*28*28 + Y*28+X]);
			}
			__syncthreads();
		}
	   	__syncthreads();

		///////////////////////////【反向传播】卷积1层---->输入层（卷积核）的权值更新///////////////////////////
		tmpSum = 0;

		for(int NumK = 0; NumK < 100; NumK++)   	
		{
			tmpSum = 0;
			if (X < 5 && Y < 5) 
			{
			 	for (int j = 0; j < 28; j++) 
				{
					for (int i = 0; i < 28; i++) 
					{
						//卷积操作一般是要把卷积核旋转180度再做协相关（一一位置对应相乘），不过，由于 CNN 中的卷积参数本来就是学出来的，所以旋不旋转，关系其实不大，这里默认不旋转
						tmpSum += pPadding_NetInput[(Y+j)*32+(X+i)] * pDelta_Convolution1[NumK*28*28 + (j*28+i)];
					}
				}
				pPD_WeightItoC1[NumK*5*5 + Y*5+X] = tmpSum;
				pWeightItoC1[NumK*5*5 + Y*5+X] = (pWeightItoC1[NumK*5*5 + Y*5+X])*OldWeightPercentage + (pWeightItoC1[NumK*5*5 + Y*5+X] - LearningRate * pPD_WeightItoC1[NumK*5*5 + Y*5+X])*NewWeightPercentage;
			}
		   	__syncthreads();
	   	}	
	   	__syncthreads();
	}
}



//*****************************************	以上为GPU上运行的代码	*****************************************//

int main(void) 
{
	// 为设备副本分配空间
	HANDLE_ERROR( hipMalloc((void **)&d_pNetInput60000,			sizeof(float)*48249054) ); //sizeof(int) = sizeof(float) = 4字节
		
	// 为主机副本分配空间
	pPicture				= (unsigned char*)malloc(N*TestNumber);
	pNetInput60000			= (float*)malloc(sizeof(float)*48249054);

	/******************** CPU上开的整块内存中的分配用于定位的指针 ********************/	
	AllocatePointer();

	//读图片和标签
	ReadTrainPicData(pPicture, pLabel60000);
//	ReadTest10kPicData(pPicture, pLabel60000);

	//读权重
	ReadWeightData(pWeightItoC1, pWeightS2toO);
//	RandomInitWeight(pWeightItoC1, pWeightS2toO);

	/******************** 强制类型转换 ********************/
	TypeCasting();

	//将输入复制到设备
	HANDLE_ERROR( hipMemcpy(d_pNetInput60000,  pNetInput60000,  sizeof(float)*48249054, 	hipMemcpyHostToDevice) );

	/******************** 进入网络训练 ********************/
	MaxCorrect = 0;
	for (int traintimes = 1; traintimes < 50; traintimes++)
	{
	 	correct = 0;
		CorrectRate = 0;
		
		printf("第%d次训练：\n", traintimes);

		RunCUDA(CheckNumber);
		HANDLE_ERROR( hipMemcpy(pNetInput60000,	d_pNetInput60000,	sizeof(float)*48249054,	hipMemcpyDeviceToHost) );

		//******************** (1)判断图片被识别为数字几(2)判断识别正误(3)分类 ********************
		for (iPicNum = 0; iPicNum < CheckNumber; iPicNum++)
		{
			pSigmoidOutput = 	pSigmoidOutput60000 + 	M*iPicNum;
			pLabel = 		pLabel60000 + 		iPicNum;

			RecognizeResult();
			JudgeRightOrWrong(iPicNum, Recognize);
		}
		CorrectRate = ((float)correct) / CheckNumber;
		printf("正确数：%d 正确率：%.4f%%\n", correct, CorrectRate * 100);
		if(correct > MaxCorrect)
		{
			MaxCorrect = correct;
			WriteWeightData(pWeightItoC1, pWeightS2toO);
		}
	}
	/******************** 释放主机与设备的内存空间 ********************/
	free(pPicture);
	free(pNetInput60000);
	HANDLE_ERROR( hipFree(d_pNetInput60000) ); 
}


//*****************************************	以下为待调用的函数	*****************************************//
/******************** CPU上开的整块内存中的指针定位分配 ********************/
void AllocatePointer()
{
	//内存中的指针分配
	pPadding_NetInput =			pNetInput60000 + 						N*TestNumber;
	pWeightItoC1 =				pPadding_NetInput + 					1024;
	pWeightS2toO =				pWeightItoC1 + 						25*100;
	pNetConvolution1 =			pWeightS2toO + 						196*M*100;
	pSigmoidConvolution1 =			pNetConvolution1 + 					N*100;
	pNetSubsampling2 =			(struct MAXPool*)(pSigmoidConvolution1 + 		N*100);
	pNetOutput =				(float*)(pNetSubsampling2 + 				196*100);
	pSigmoidOutput60000 =			pNetOutput + 						M;
	pTarget =					pSigmoidOutput60000 + 					M*TestNumber;			
	pDelta_NetOutput =			pTarget + 							M;	
	pDelta_Subsampling2 =			pDelta_NetOutput + 					M;
	pUpsample_Delta_Subsampling2 =		pDelta_Subsampling2 + 					196*100;	
	pDelta_Convolution1 =			pUpsample_Delta_Subsampling2 +				N*100;				
	pPD_WeightItoC1 =				pDelta_Convolution1 + 					N*100;	
	pLabel60000 =				(unsigned char*)(pPD_WeightItoC1 + 			25*100);

}

/******************** 启用CUDA ********************/
void RunCUDA(int iCheck)
{
	//防止全局指针pNetInput的值一直在累加，跑出数据的有效范围
	pNetInput = 0;
	pSigmoidOutput = 0;
	pLabel = 0;

	for(int iNum = 0; iNum < iCheck; iNum += GPUcalculateNumber)
	{
		HANDLE_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(d_iNum), &iNum, sizeof(int)) );

		ForwardAndBackPropagation<<<1, threadsPerBlock>>>(d_pNetInput60000, d_pNetInput, d_pPadding_NetInput, d_pWeightItoC1, d_pNetConvolution1, d_pSigmoidConvolution1, d_pNetSubsampling2, d_pWeightS2toO, d_pNetOutput, d_pSigmoidOutput60000, d_pSigmoidOutput, d_pTarget, d_pDelta_NetOutput, d_pDelta_Subsampling2, d_pUpsample_Delta_Subsampling2, d_pDelta_Convolution1, d_pPD_WeightItoC1, d_pLabel60000, d_pLabel);
	}
}

/******************** 判断图片是否被正确识别为对应的label ********************/
void RecognizeResult()
{
	for(int i = 0; i < 10; i++)
	{
		//通过比较SigmoidOutput的大小，得到的最大值即为认定图片被网络识别成几，再与Label比较，看是否识别正确		
		if(*(pSigmoidOutput + i) > *(pSigmoidOutput + Recognize))
		{
			Recognize = i;
		}
	}
}

/******************** 判断图片识别正确或错误 ********************/
void JudgeRightOrWrong(int iNum, int OCR)
{
	if(OCR == *pLabel)
	{
		correct++;
//		printf("第%d张图片被正确识别为%d\n", iNum, *pLabel);
	}
	else
	{
//		printf("第%d张图片应为%d被错误识别为%d\n", iNum, *pLabel, OCR);
	}
	
}



//************************************************************************************************************************
/******************** 读【训练集】图片文件 ********************/
int ReadTrainPicData(unsigned char*data, unsigned char*pLabel60000)
{
	int iCheck = 0;

	FILE *fp_image = fopen("MNIST/train-images.idx", "rb");//打开这个二进制(b)文件train-images.idx，以只读(r)的方式打开，
	FILE *fp_label = fopen("MNIST/train-labels.idx", "rb");//读类标
	//FILE *fp_image = fopen("MNIST/t10k_images.idx", "rb");//打开这个二进制(b)文件t10k-images.idx，以只读(r)的方式打开，
	//FILE *fp_label = fopen("MNIST/t10k_labels.idx", "rb");//读类标
	if (!fp_image || !fp_label) 
		return 1;

	fseek(fp_image, 16, SEEK_SET);//从文件开始位置偏移16字节，定位像素数据开始
	iCheck = fread(data, sizeof(char)*pixels*pixels, TestNumber, fp_image);//从文件(fp_image)中读(pixels * pixels)字节数，读TestNumber次，到data指向的内存中
	if (iCheck != TestNumber)
	{
		printf("读图片文件失败！\n");
	}
	else
	{
		printf("读图片文件成功！\n");	
	}

	fseek(fp_label, 8, SEEK_SET);//从文件开始位置偏移8字节，定位标签数据
	iCheck = fread(pLabel60000, sizeof(char), TestNumber, fp_label);//从文件(fp_label)中读sizeof(char)个字节数，读TestNumber次，读到pLabel指向的内存中
	if (iCheck != TestNumber)
	{
		printf("读Label文件失败！\n");
	}
	else
	{
		printf("读Label文件成功！\n");	
	}

	fclose(fp_image);
	fclose(fp_label);

	return 0;
}

/******************** 读【测试集】图片文件 ********************/
int ReadTest10kPicData(unsigned char*data, unsigned char*pLabel60000)
{
	int iCheck = 0;

	FILE *fp_image = fopen("MNIST/t10k_images.idx", "rb");//打开这个二进制(b)文件t10k-images.idx，以只读(r)的方式打开，
	FILE *fp_label = fopen("MNIST/t10k_labels.idx", "rb");//读类标
	if (!fp_image || !fp_label) 
		return 1;

	fseek(fp_image, 16, SEEK_SET);//从文件开始位置偏移16字节，定位像素数据开始
	iCheck = fread(data, sizeof(char)*pixels*pixels, CheckNumber, fp_image);//从文件(fp_image)中读(pixels * pixels)字节数，读CheckNumber次，到data指向的内存中
	if (iCheck != CheckNumber)
	{
		printf("读图片文件失败！\n");
	}
	else
	{
		printf("读图片文件成功！\n");	
	}

	fseek(fp_label, 8, SEEK_SET);//从文件开始位置偏移8字节，定位标签数据
	iCheck = fread(pLabel60000, sizeof(char), CheckNumber, fp_label);//从文件(fp_label)中读sizeof(char)个字节数，读CheckNumber次，读到pLabel指向的内存中
	if (iCheck != CheckNumber)
	{
		printf("读Label文件失败！\n");
	}
	else
	{
		printf("读Label文件成功！\n");	
	}

	fclose(fp_image);
	fclose(fp_label);

	return 0;
}


/******************** 读文件【训好的权重】 ********************/
void ReadWeightData(float *WeightItoC1, float *WeightS2toO)
{
	int iNum = 0; //神经元个数

	FILE *fpReadWeightItoC1 = fopen("MNIST01/fWeightItoC1Data201811102304.wet","rb");
	FILE *fpReadWeightS2toO = fopen("MNIST01/fWeightS2toOData201811102304.wet","rb");
	
	if(fpReadWeightItoC1 == NULL)
	{
		printf("fWeightItoC1Data201811102304.wet文件不存在！\n");
		return;
	}
	if(fpReadWeightS2toO == NULL)
	{
		printf("fWeightS2toOData201811102304.wet文件不存在！\n");
		return;
	}

	//下面是读数据，将读到的数据存到数组WeightItoC1[NeuronNumConvolution1]中
	iNum = fread(WeightItoC1, sizeof(float), 5*5*100, fpReadWeightItoC1);
	if(iNum != 5*5*100)
	{
		printf("读数据存到数组WeightItoC1[5*5*100]错误！！！\n");
	}
	printf("读【输入层-卷积1层】间的权重数据成功\n");

	//下面是读数据，将读到的数据存到数组WeightS2toO[14*14*100*10]中
	iNum = fread(WeightS2toO, sizeof(float), 14*14*100*10, fpReadWeightS2toO);
	if(iNum != 14*14*100*10)
	{
		printf("读数据存到数组WeightS2toO[14*14*100*10]错误！！！\n");
	}
	printf("读【池化2层-输出层】间的权重数据成功\n");

	fclose(fpReadWeightItoC1);
	fclose(fpReadWeightS2toO);

	return;
}

/******************** 写文件【训好的权重】 ********************/
void WriteWeightData(float *WeightItoC1, float *WeightS2toO)
{
	int iNum = 0;
	FILE *fpWriteWeightItoC1 = fopen("MNIST02/fWeightItoC1Data201811102304.wet","wb+");
	FILE *fpWriteWeightS2toO = fopen("MNIST02/fWeightS2toOData201811102304.wet","wb+");
	
	if(fpWriteWeightItoC1 == NULL)
	{
		printf("创建WeightItoC1Data201811102304.wet文件失败！\n");
		return;
	}
	if(fpWriteWeightS2toO == NULL)
	{
		printf("创建WeightS2toOData201811102304.wet文件失败！\n");
		return;
	}

	//将输入层-卷积1层间的权重写入到WeightItoC1Data201811102304.wet文件中
	iNum = fwrite(WeightItoC1, sizeof(float), 5*5*100, fpWriteWeightItoC1);
	if (iNum != 5*5*100)
	{
		printf("写WeightItoC1Data201811102304.wet文件失败！");
	}
	printf("所有【输入层-卷积1层】间的权重数据已经保存\n");
	//将池化2层-输出层间的权重写入到WeightS2toOData201811102304.wet文件中
	iNum = fwrite(WeightS2toO, sizeof(float), 14*14*100*10, fpWriteWeightS2toO);
	if (iNum != 14*14*100*10)
	{		
		printf("写WeightS2toOData201811102304.wet文件失败！");
	}
	printf("所有【池化2层-输出层】间的权重数据已经保存\n");

	fclose(fpWriteWeightItoC1);
	fclose(fpWriteWeightS2toO);
}


//随机初始化权重
void RandomInitWeight(float *WeightItoC1, float *WeightS2toO)
{
	for(int m = 0; m < 5*5*100; m++)
	{
		WeightItoC1[m] = (rand()/(float)(RAND_MAX))/1000; //【输入层-卷积1层】之间任意2个神经元间的权重W
	}
	printf("所有【输入层-卷积1层】间的权重数据已经初始化完毕！\n");

	for(int m = 0; m < 14*14*100*10; m++)
	{
		WeightS2toO[m] = (rand()/(float)(RAND_MAX))/20000; //【池化2层-输出层】之间任意2个神经元间的权重W
	}
	printf("所有【池化2层-输出层】间的权重数据已经初始化完毕！\n");
}

/******************** 强制类型转换 ********************/
void TypeCasting()
{
	for(int iNum = 0; iNum < N*TestNumber; iNum++)
	{
		*(pNetInput60000 + iNum) = (float)(*(pPicture + iNum));
	}
}

